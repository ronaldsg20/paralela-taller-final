#include "hip/hip_runtime.h"
//BLOCKWISE
#include "omp.h"
#include <stdio.h>
#include <stdlib.h>
//For  CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>


using namespace std;

__global__  void Multiply(int ** a, int ** b, int ** c, int ID) {
    
    int ini = (int)(SIZE/omp_get_num_threads())*ID;
	int fin = (int)(SIZE/omp_get_num_threads())+ini;
	for (int i = ini ; i < fin ;i++ ) {
		for (int j = 0 ; j < SIZE ; j++) {
			for (int k = 0; k < SIZE; k++) {
				c[i][j] += a[i][k] * b[k][j];
			}
		}
	}
}



int main(int argc, char **argv)
{
    //define variables
    int  n  = 16;
    //Host matrix
    int* h_a;
    int* h_b;
    int* h_c;

    //Device  matrix
    int* d_a;
    int* d_b;
    int* d_c;

    size_t  bytes = n*n*sizeof(int);

    //Allocate memory in host
    h_a =(int*)malloc(bytes);
    h_b =(int*)malloc(bytes);
    h_c =(int*)malloc(bytes);

    //Initialize matrix
    for (int i=0;i<n;i++){
        for(int  j=0;j<n;j++){
            h_a[i*n+j]  =  rand()%10;
            h_b[i*n+j]  =  rand()%10;
            h_c[i*n+j] =  0;
        }
    }
    
    
    return 0;

}
