//BLOCKWISE

#include <hip/hip_runtime.h>
#include <iostream>
#include "omp.h"


using namespace std;
//global variables
int  SIZE,THREADS;

int ** randomMatrix () {
	int **result = new int *[SIZE] ;

	for (int i = 0; i < SIZE; i++) {
		result[i] = new int [SIZE] ;

		for (int j = 0; j < SIZE; j++)
			result[i][j] = rand()%6;
	}

	return result;
}

void Multiply(int ** a, int ** b, int ** c, int ID) {
    
    int ini = (int)(SIZE/omp_get_num_threads())*ID;
	int fin = (int)(SIZE/omp_get_num_threads())+ini;
	for (int i = ini ; i < fin ;i++ ) {
		for (int j = 0 ; j < SIZE ; j++) {
			for (int k = 0; k < SIZE; k++) {
				c[i][j] += a[i][k] * b[k][j];
			}
		}
	}
}



int main(int argc, char **argv)
{
     // read arguments
    if ( argc != 3 )
    {
        printf("usage: ./matrix  <SIZE> <THREADS>\n");
        return -1;
    }
    
    SIZE =atoi(argv[1]);
    THREADS = atoi(argv[2]);

   
    int **a=new int*[SIZE];
    int **b=new int*[SIZE];
    int **c=new int*[SIZE];

    a=randomMatrix();
    b=randomMatrix();

    for (int i = 0; i < SIZE; i++) {
		c[i] = new int [SIZE] ;

		for (int j = 0; j < SIZE; j++){
            c[i][j] = 0;
        }
			
	}

   #pragma omp parallel num_threads(THREADS)
	{
		int ID = omp_get_thread_num();
		Multiply(a, b, c, ID);
	}
   // Displaying the multiplication of two matrix.
   cout << endl << "Output Matrix: " << endl;
    for(int i = 0; i <SIZE; ++i)
    for(int j = 0; j < SIZE; ++j)
    {
        cout << " " << a[i][j];
        if(j == SIZE-1)
            cout << endl;
    }

    cout<<endl;
    for(int i = 0; i <SIZE; ++i)
    for(int j = 0; j < SIZE; ++j)
    {
        cout << " " << b[i][j];
        if(j == SIZE-1)
            cout << endl;
    }
    cout<<endl;
    for(int i = 0; i <SIZE; ++i)
    for(int j = 0; j < SIZE; ++j)
    {
        cout << " " << c[i][j];
        if(j == SIZE-1)
            cout << endl;
    }
    cout<<endl; 
  
    
    
    return 0;

}
