#include "hip/hip_runtime.h"
//BLOCKWISE
#include "omp.h"
#include <stdio.h>
#include <stdlib.h>
//For  CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void multiplication(int*  a,int* b,int*  c,int n){
    //Calculate row and column
    //int r =blockIdx.y*blockDim.y+threadIdx.y
    int row = (blockDim.y * blockIdx.y) + threadIdx.y;
    int col= (blockDim.x * blockIdx.x) + threadIdx.x;
    int partial=0;

    for(int i=0;i<n;i++){
        partial  += a[row * n +i] * b[i*n+col];
    }

    c[row*n+col]=partial;
}



int main(int argc, char **argv)
{
    printf("Inicio");
    //define variables
    int  n  = 1024;
    //Host matrix
    int* h_a;
    int* h_b;
    int* h_c;
    int* h_c2;

    //Device  matrix
    int* d_a;
    int* d_b;
    int* d_c;

    size_t  bytes = n*n*sizeof(int);

    //Allocate memory in host
    h_a =(int*)malloc(bytes);
    h_b =(int*)malloc(bytes);
    h_c =(int*)malloc(bytes);
    h_c2 =(int*)malloc(bytes);

    //Initialize matrix
    for (int i=0;i<n;i++){
        for(int  j=0;j<n;j++){
            h_a[i*n+j]  =  rand()%10;
            h_b[i*n+j]  =  rand()%10;
        }
    }

    //Allocate memory  in device
    hipMalloc(&d_a,bytes);
    hipMalloc(&d_b,bytes);
    hipMalloc(&d_c,bytes);

    //Copy  data  host to   device
    hipMemcpy(d_a,h_a,bytes,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,bytes,hipMemcpyHostToDevice);

    //Write blocks  and threads
    int threads_block =  16;
    dim3 block_size(threads_block,threads_block);
    dim3 grid_size(n/block_size.x,n/block_size.y);
    printf("Aqui");
      //<<<Bloques,hilos>>>
    multiplication  <<<grid_size,block_size>>> (d_a,d_b,d_c,n);
   
    //Copy  data  device to host
    printf("hola1");
    hipMemcpy(h_c,d_c,bytes,hipMemcpyDeviceToHost);
    printf("hola2");
    printf("%d %d\n",h_c[0],h_c2[0]);
    // free memory

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    
    
    return 0;

}
