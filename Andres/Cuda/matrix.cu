#include "hip/hip_runtime.h"
//BLOCKWISE
#include "omp.h"
#include <stdio.h>
#include <stdlib.h>
//For  CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void multiplication(int*  a,int* b,int*  c,int n){
    //Calculate row and column
    //int r =blockIdx.y*blockDim.y+threadIdx.y
    int row = (blockDim.y * blockIdx.y) + threadIdx.y;
    int col= (blockDim.x * blockIdx.x) + threadIdx.x;
    int partial=0;

    for(int i=0;i<n;i++){
        partial  += a[row * n +i] * b[i*n+col];
    }

    c[row*n+col]=partial;
}
/*__host__  void  multiplication2(int* a,int* b,int* c,int  size){
    for(int i=0;i<size;i++){
        for(int   j=0;j<size;i++){
            for(int k=0;k<size;k++){
                c[i*size+j]  += a[i*size+k] * b[k*size+j];
            }
        }
    }
}*/


int main(int argc, char **argv)
{
    
    //define variables
    int  n  = 1024;
    //Host matrix
    int* h_a;
    int* h_b;
    int* h_c;
    //int* h_c_s;

    //Device  matrix
    int* d_a;
    int* d_b;
    int* d_c;

    size_t  bytes = n*n*sizeof(int);

    //Allocate memory in host
    h_a =(int*)malloc(bytes);
    h_b =(int*)malloc(bytes);
    h_c =(int*)malloc(bytes);
   // h_c_s =(int*)malloc(bytes);

    //Initialize matrix
    for (int i=0;i<n;i++){
        for(int  j=0;j<n;j++){
            h_a[i*n+j]  =  rand()%10;
            h_b[i*n+j]  =  rand()%10;
        }
    }

    //Allocate memory  in device
    hipMalloc(&d_a,bytes);
    hipMalloc(&d_b,bytes);
    hipMalloc(&d_c,bytes);

    //Copy  data  host to   device
    hipMemcpy(d_a,h_a,bytes,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,bytes,hipMemcpyHostToDevice);

    //Write blocks  and threads
    int threads_block =  2;
    dim3 block_size(threads_block,threads_block);
    dim3 grid_size(n/block_size.x,n/block_size.y);
      //<<<Bloques,hilos>>>
    
    multiplication  <<<grid_size,block_size>>> (d_a,d_b,d_c,n);
    //multiplication2(h_a,h_b,h_c_s,n);
  
    //Copy  data  device to host
  
    hipMemcpy(h_c,d_c,bytes,hipMemcpyDeviceToHost);
 
    printf( "Output Matrix: \n");
    for(int i = 0; i <n; ++i)
    for(int j = 0; j < n; ++j)
    {
        printf(" %d",h_a[n*i+j]) ;
        if(j == n-1)
            printf("\n");
    }

    printf("\n");
    for(int i = 0; i <n; ++i)
    for(int j = 0; j < n; ++j)
    {
        printf(" %d",h_b[n*i+j]) ;
        if(j == n-1)
           printf("\n");
    }
    printf("\n");
    for(int i = 0; i <n; ++i)
    for(int j = 0; j < n; ++j)
    {
        printf(" %d",h_c[n*i+j]) ;
        if(j == n-1)
           printf("\n");
    }
    printf("\n");
    // free memory

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    //free(h_c_s);

    
    
    return 0;

}
