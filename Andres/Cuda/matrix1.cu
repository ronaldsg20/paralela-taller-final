#include "hip/hip_runtime.h"
//BLOCKWISE
#include "omp.h"
#include <stdio.h>
#include <stdlib.h>
//For  CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void multiplication(int*  a,int* b,int*  c,int n){
    //Calculate row and column
    //int r =blockIdx.y*blockDim.y+threadIdx.y
    int tn = (blockDim.x * blockIdx.x) + threadIdx.x;
    
    int ini = n/(int)*block_size)*(tn);
    int fin = n/(int)*block_size)+ini;

    int i, j, k; 
    if(tn <n){
        for (i = ini; i < fin; i++) { 
            for (j = 0; j < n; j++) { 
                for (k = 0; k < n; k++) 
                    c[i][j] += a[i][k]*b[k][j]; 
            } 
        }
    }
}
/*__host__  void  multiplication2(int* a,int* b,int* c,int  size){
    for(int i=0;i<size;i++){
        for(int   j=0;j<size;i++){
            for(int k=0;k<size;k++){
                c[i*size+j]  += a[i*size+k] * b[k*size+j];
            }
        }
    }
}*/


int main(int argc, char **argv)
{
    
    //define variables
    int  n  = atoi(argv[1]);
    int threads_block =  atoi(argv[2]);
    //Host matrix
    int* h_a;
    int* h_b;
    int* h_c;
    //int* h_c_s;

    //Device  matrix
    int* d_a;
    int* d_b;
    int* d_c;

    size_t  bytes = n*n*sizeof(int);

    //Allocate memory in host
    h_a =(int*)malloc(bytes);
    h_b =(int*)malloc(bytes);
    h_c =(int*)malloc(bytes);
   // h_c_s =(int*)malloc(bytes);

    //Initialize matrix
    for (int i=0;i<n;i++){
        for(int  j=0;j<n;j++){
            h_a[i*n+j]  =  rand()%10;
            h_b[i*n+j]  =  rand()%10;
        }
    }

    //Allocate memory  in device
    hipMalloc(&d_a,bytes);
    hipMalloc(&d_b,bytes);
    hipMalloc(&d_c,bytes);

    //Copy  data  host to   device
    hipMemcpy(d_a,h_a,bytes,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,bytes,hipMemcpyHostToDevice);

    //Write blocks  and threads
    
    dim3 block_size(threads_block,threads_block);
    dim3 grid_size(n/block_size.x,n/block_size.y);
      //<<<Bloques,hilos>>>
    
    multiplication  <<<grid_size,block_size>>> (d_a,d_b,d_c,n);
    //multiplication2(h_a,h_b,h_c_s,n);
  
    //Copy  data  device to host
  
    hipMemcpy(h_c,d_c,bytes,hipMemcpyDeviceToHost);
 
    /*printf( "Output Matrix: \n");
    for(int i = 0; i <n; ++i)
    for(int j = 0; j < n; ++j)
    {
        printf(" %d",h_a[n*i+j]) ;
        if(j == n-1)
            printf("\n");
    }

    printf("\n");
    for(int i = 0; i <n; ++i)
    for(int j = 0; j < n; ++j)
    {
        printf(" %d",h_b[n*i+j]) ;
        if(j == n-1)
           printf("\n");
    }
    printf("\n");
    for(int i = 0; i <n; ++i)
    for(int j = 0; j < n; ++j)
    {
        printf(" %d",h_c[n*i+j]) ;
        if(j == n-1)
           printf("\n");
    }
    printf("\n");*/
    // free memory

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    //free(h_c_s);

    
    
    return 0;

}
