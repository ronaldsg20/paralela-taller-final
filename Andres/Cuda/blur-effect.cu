#include "hip/hip_runtime.h"

/**
 * Blur-effect
 */

 #include <stdio.h>
 #include <math.h>
 // For the CUDA runtime routines (prefixed with "cuda_")
 #include <hip/hip_runtime.h>
 #include <hip/hip_runtime_api.h>
 #include <hip/hip_runtime.h>

 #include <stdlib.h>
 #include <cstdint>
 #include <opencv2/opencv.hpp>

//#include <cuPrintf.cuh>
//#include "cuPrintf.cu"
 
  using namespace cv;
  using namespace std;
  

// function aviable only on the device

  __device__ void aplyBlur(int x, int y, int *kernel,int *w, int *h, int *input, int *output){
    // collect the average data of neighbours 
    int blue,green,red;
    blue=green=red=0;
    int n=0;
    int pixel_pos;
    int k= (int)*kernel;
    //int wt = (int)*w;

    for(int i = x - (k/2); i < x+(k/2); i++)
    {    
        for (int j = y-(k/2); j < y+(k/2); j++)
        {
            //check if the point is in the image limits
            if(0<=i && i<((int)*w)-1 && 0<=j && j<((int)*h)-1){
                pixel_pos = (j*((int)*w)*3)+(i*3);
                blue += input[pixel_pos+0];
                green += input[pixel_pos+1];
                red += input[pixel_pos+2];
                n++;
            }
        }
    }
    pixel_pos = (y*((int)*w)*3)+(x*3);
    if(n!=0){
         //write the average on the output image
        output[pixel_pos+0]=(blue/n);
        output[pixel_pos+1]=(green/n);
        output[pixel_pos+2]=(red/n);
    }
   
}

 /**
  * CUDA Kernel Device code
  * 
  */ 
 /*****************************************************************************/
 
 __global__ void blur(int *input,int *output, int *kernel, int *totalThreads, int *width, int *height)
 {   
     
    int tn = (blockDim.x * blockIdx.x) + threadIdx.x;
    
    int ini = (int)((int)*width/(int)*totalThreads)*(tn);
    int fin = (int)((int)*width/(int)*totalThreads)+ini;
    if(tn<*width){
        for (int i = ini; i < fin; i++)
        {
            for (int j = 0; j < (int)*height; j++)
            {
            aplyBlur(i,j,kernel, width, height,input, output);


            }
        }
    }
    
     
 }
 
 
 /******************************************************************************
  * Host main routine
  */
 int main(int argc, char **argv)
 {   
     // define variables
     int h_threads;
     int h_kernel;
     int h_width;
     int h_height;

     int *d_threads;
     int *d_kernel;
     int *d_width;
     int *d_height;

     Mat output;
     Mat input;
    // handle errors

    hipError_t error = hipSuccess;
     //********************read parameters**********************
     if ( argc != 4 )
    {
        printf("usage: ./blur-effect <Image_Path> <Image_out_Path> <KERNEL>n");
        return -1;
    }

    String oFile = argv[2];

    //read the image and set width and height
    input = imread( argv[1], IMREAD_COLOR );
    
    if ( !input.data )
    {
        printf("No image data \n");
        return -1;
    }
    h_width = input.rows;
    h_height =input.cols;
    
    h_kernel = atoi(argv[3]);

    // define the output as a clone of input image
    output = input.clone();
    //imwrite( oFile, output ); // just for test


    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);

   int blocks = deviceProp.multiProcessorCount;

   h_threads = h_width/blocks;


    printf(" Processing image %s \n width: %d  - Heigh : %d \n",argv[1],h_width,h_height);

    // ************************ image pointers ***********************************
    int *d_input;
    int *d_output;
    int *h_input;
    int *h_output;

   hipMalloc(&d_height,sizeof(int));
   hipMalloc(&d_kernel,sizeof(int));
   hipMalloc(&d_width,sizeof(int));
   hipMalloc(&d_threads,sizeof(int));
   hipMalloc(&d_input,h_width*h_height*sizeof(int)*3);
   hipMalloc(&d_output,h_width*h_height*sizeof(int)*3);
    
     size_t size = h_width * h_height * 3 * sizeof(int);
     h_input = (int *)malloc(size);
     h_output = (int *)malloc(size);

     // set initial values
     Vec3b pixel;
     //pixel = input.at<Vec3b>(Point(i,j));
     printf("ORIGINAL IMAGE \n");
     for(int i=0;i<h_width;i++){
       for(int j=0;j<h_height;j++){
        
        h_input[(j*h_width*3)+(i*3)+0]= input.at<Vec3b>(i,j)[0];
        h_input[(j*h_width*3)+(i*3)+1]= input.at<Vec3b>(i,j)[1];
        h_input[(j*h_width*3)+(i*3)+2]= input.at<Vec3b>(i,j)[2];
       }
     }

     // MemCpy: host to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, &h_kernel, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_threads, &h_threads, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_width, &h_width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_height, &h_height, sizeof(int), hipMemcpyHostToDevice);
    
    printf("CudaMemcpy host to device done.\n");

     // Launch kernel 
     blur<<<blocks,h_threads>>>(d_input,d_output, d_kernel, d_threads, d_width, d_height);

     // MemCpy: device to host
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
     for(int i=0;i<h_width;i++){
       for(int j=0;j<h_height;j++){
        
        output.at<Vec3b>(i, j)[0] = h_output[(j*h_width*3) + (i * 3) + 0];
        output.at<Vec3b>(i, j)[1] = h_output[(j*h_width*3) + (i * 3) + 1];
        output.at<Vec3b>(i, j)[2] = h_output[(j*h_width*3) + (i * 3) + 2];
       }
     }

     // save data
    
     try {
        imwrite( oFile, output );
    }
    catch (runtime_error& ex) {
        fprintf(stderr, "Exception saving image : %s\n", ex.what());
        return 1;
    }

     // free memory

     hipFree(d_height);
     hipFree(d_width);
     hipFree(d_output);
     hipFree(d_input);
     hipFree(d_kernel);
     hipFree(d_threads);

     free(h_input);
     free(h_output);

     return 0;
 }
 
 