#include "hip/hip_runtime.h"
//BLOCKWISE
#include "omp.h"
#include <stdio.h>
#include <stdlib.h>
//For  CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void multiplication(int*  a,int* b,int*  c,int n,int threads){
    //Calculate row and column
    //int r =blockIdx.y*blockDim.y+threadIdx.y
    int tn = (blockDim.x * blockIdx.x) + threadIdx.x;
    
    int ini = n/threads*(tn);
    int fin = n/threads+ini;

    int i, j, k; 
    if(tn <n){
        for (i = ini; i < fin; i++) { 
            for (j = 0; j < n; j++) { 
                for (k = 0; k < n; k++) 
                    c[i*n+j]  += a[i*n+k] * b[k*n+j];
            } 
        }
    }
}
/*__host__  void  multiplication2(int* a,int* b,int* c,int  size){
    for(int i=0;i<size;i++){
        for(int   j=0;j<size;i++){
            for(int k=0;k<size;k++){
                c[i*size+j]  += a[i*size+k] * b[k*size+j];
            }
        }
    }
}*/


int main(int argc, char **argv)
{
    
    //define variables
    int  n  = atoi(argv[1]);

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);

    int blocks = deviceProp.multiProcessorCount;
    int threads= (int)(n/blocks);
    //Host matrix
    int* h_a;
    int* h_b;
    int* h_c;
    //int* h_c_s;

    //Device  matrix
    int* d_a;
    int* d_b;
    int* d_c;

    size_t  bytes = n*n*sizeof(int);

    //Allocate memory in host
    h_a =(int*)malloc(bytes);
    h_b =(int*)malloc(bytes);
    h_c =(int*)malloc(bytes);
   // h_c_s =(int*)malloc(bytes);

    //Initialize matrix
    for (int i=0;i<n;i++){
        for(int  j=0;j<n;j++){
            h_a[i*n+j]  =  rand()%10;
            h_b[i*n+j]  =  rand()%10;
        }
    }

    //Allocate memory  in device
    hipMalloc(&d_a,bytes);
    hipMalloc(&d_b,bytes);
    hipMalloc(&d_c,bytes);

    //Copy  data  host to   device
    hipMemcpy(d_a,h_a,bytes,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,bytes,hipMemcpyHostToDevice);

    //Write blocks  and threads
    
    //dim3 block_size(threads_block,threads_block);
      //<<<Bloques,hilos>>>
    
    multiplication  <<<blocks,threads>>> (d_a,d_b,d_c,n,threads);
    //multiplication2(h_a,h_b,h_c_s,n);
  
    //Copy  data  device to host
  
    hipMemcpy(h_c,d_c,bytes,hipMemcpyDeviceToHost);
 
   if (n<9){
    printf( "Output Matrix: \n");
    for(int i = 0; i <n; ++i)
    for(int j = 0; j < n; ++j)
    {
        printf(" %d",h_a[n*i+j]) ;
        if(j == n-1)
            printf("\n");
    }

    printf("\n");
    for(int i = 0; i <n; ++i)
    for(int j = 0; j < n; ++j)
    {
        printf(" %d",h_b[n*i+j]) ;
        if(j == n-1)
           printf("\n");
    }
    printf("\n");
    for(int i = 0; i <n; ++i)
    for(int j = 0; j < n; ++j)
    {
        printf(" %d",h_c[n*i+j]) ;
        if(j == n-1)
           printf("\n");
    }
    printf("\n");

   }
    
    // free memory

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    //free(h_c_s);

    
    
    return 0;

}
