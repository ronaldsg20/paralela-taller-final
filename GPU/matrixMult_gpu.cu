#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <omp.h>
#include <stdio.h>
#include <string.h>
 #include <math.h>
 // For the CUDA runtime routines (prefixed with "cuda_")
 #include <hip/hip_runtime.h>
 #include <hip/hip_runtime_api.h>
 #include <hip/hip_runtime.h>
// Global variables


void readMatrix(char *filename, int M[][1024], int N){
    FILE *fstream = fopen(filename, "r");
    if(fstream == NULL){
        printf("\n file opening failed ");
        return;
    }
    char *record,*line;
    char buffer[1024];
    int i=0,j=0;
    while((line=fgets(buffer,sizeof(buffer),fstream))!=NULL){
        j = 0;
        record = strtok(line,",");
        while(record != NULL){
            M[i][j++] = atoi(record);
            record = strtok(NULL,",");
        }
        ++i;
    }
}

void printMatrix(int M[][1024], int N){
    // print matrix for testing
    int i;
    int j;
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%d ",M[i][j]);
        }
        printf("\n");
    }
    printf("\n");
}

void multiplyMatrix(int A[][1024], int B[][1024], int C[][1024], int ini, int fin) 
{ 
    int i, j, k; 
    for (i = ini; i < fin; i++) { 
        for (j = 0; j < 1024; j++) { 
            C[i][j] = 0; 
            for (k = 0; k < 1024; k++) 
                C[i][j] += A[i][k]*B[k][j]; 
        } 
    } 
}

void writeMatrix(char *filename, int R[][1024], int N){
    FILE *fp;
    int i,j;
    fp=fopen(filename,"w+");
    for(i=0 ; i<N; i++){
        for(j=0; j<N; j++){
            fprintf(fp,",%d ",R[i][j]);
        }
        fprintf(fp,"\n%d",i+1);
    }
    fclose(fp);
}

__global__ void multiplyMat(int *N,int *H){
    int tn = (blockDim.x * blockIdx.x) + threadIdx.x;
    
    int ini = (int)((int)*N/(int)*H)*(tn);
    int fin = (int)((int)*N/(int)*H)+ini;

    int i, j, k; 
    if(tn <*N){
        for (i = ini; i < fin; i++) { 
            for (j = 0; j < 1024; j++) { 
                C[i][j] = 0; 
                for (k = 0; k < 1024; k++) 
                    C[i][j] += A[i][k]*B[k][j]; 
            } 
        }
    }
    

}

int main(int argc, char **argv){

    int A[1024][1024];
    int B[1024][1024];
    int C[1024][1024];
    //Handle errors
    hipError_t error = hipSuccess;

    // Arguments
    if ( argc !=  5){
        printf("usage: ./matrixMult_gpu MatA.csv MatB.csv N H\n");
        return -1;
    }
    char* fileA = argv[1];
    char* fileB = argv[2];
    int N = atoi(argv[3]);
    int H = atoi(argv[4]);    

    //device variables

    int *d_N;
    int *d_H;
    int d_C[1024][1024];
    int d_A[1024][1024];
    int d_B[1024][1024];

    // Read matrix A and B
    readMatrix(fileA, A, N);
    readMatrix(fileB, B, N);

    // Print matrix A and B
    printMatrix(A, N);
    printMatrix(B, N);
    
    // GPU data
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);

    // malloc and hipMalloc


    //set initial values

    //Memcpy: Host to device

    //Launch Kernel

    /* #pragma omp parallel num_threads(H)
    {
        int tn = omp_get_thread_num();
        int ini = (int)(N/H)*(tn);
        int fin = (int)(N/H)+ini;
        multiplyMatrix(A, B, C,ini,fin);
    } */ 

    // Memcpy : Device to Host

    //print results
    printMatrix(C, N);

    // Write the matrix
    writeMatrix("../Resultados/result.csv", C, N);
    
    return 0;
}