#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <omp.h>
#include <stdio.h>
#include <string.h>
 #include <math.h>
 // For the CUDA runtime routines (prefixed with "cuda_")
 #include <hip/hip_runtime.h>
 #include <hip/hip_runtime_api.h>
 #include <hip/hip_runtime.h>
// Global variables

void readMatrix(char *filename, int **M, int N){
    FILE *fstream = fopen(filename, "r");
    if(fstream == NULL){
        printf("\n file opening failed ");
        return;
    } 
    char *record,*line;
    char buffer[2300];
    int i=0,j=0;
    while((line=fgets(buffer,sizeof(buffer),fstream))!=NULL){
        j = 0;
        record = strtok(line,",");
        while(record != NULL){
            M[i][j++] = atoi(record);
            record = strtok(NULL,",");
        }
        ++i;
    }
}

void printMatrix(int **M, int N){
    // print matrix for testing
    int i;
    int j;
    for(i = 0; i < N; i++){
        for(j = 0; j < N; j++){
            printf("%d ",M[i][j]);
        }
        printf("\n");
    }
    printf("\n");
}



void writeMatrix(char *filename, int **R, int N){
    FILE *fp;
    int i,j;
    fp=fopen(filename,"w+");
    for(i=0 ; i<N; i++){
        for(j=0; j<N; j++){
            fprintf(fp,",%d ",R[i][j]);
        }
        fprintf(fp,"\n%d",i+1);
    }
    fclose(fp);
}

__global__ void multiplyMat(int *A,int *B, int *C,int *H,int *N){
    int tn,ini,fin;
    tn = (blockDim.x * blockIdx.x) + threadIdx.x;

    if(*H<=*N){
        ini = (int)((int)*N/(int)*H)*(tn);
        fin = (int)((int)*N/(int)*H)+ini;
    }else{
        ini = tn;
        fin = tn+1;
    }
    printf("Thread : %d - ini: %d - fin: %d \n",tn,ini,fin);
    int i, j, k; 
    if(tn <*N){
        for (i = ini; i < fin; i++) { 
            for (j = 0; j < *N; j++) { 
                C[(i * *N) +j] = 0; 
                for (k = 0; k < *N; k++)
                    //if(tn==0) printf("multiplying %d  with  %d \n",A[(i * *N) +k],B[(k * *N) +j]);
                    C[(i * *N) +j] += A[(i * *N) +k]*B[(k * *N) +j]; 
            } 
        }
    }
    

}

int main(int argc, char **argv){

    int **C;
    int **A;
    int **B;
    int *h_A;
    int *h_B;
    int *h_C;
    //Handle errors
    hipError_t error = hipSuccess;

    // Arguments
    if ( argc !=  7){
        printf("usage: ./matrixMult_gpu <MatA.csv> <MatB.csv> <N> <THREADSxBLOCK> <BLOCKS> <PATH-TO-MatC.csv> \n");
        return -1;
    }
    char* fileA = argv[1];
    char* fileB = argv[2];
    int N = atoi(argv[3]);
    int H = atoi(argv[4]);    

    //device variables

    int *d_N;
    int *d_H;
    int *d_C;
    int *d_A;
    int *d_B;
  
    // GPU data
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);

    // malloc and hipMalloc
    A = (int **)malloc(N * sizeof(int*));
    for(int i = 0; i < N; i++) A[i] = (int *)malloc(N * sizeof(int));
    B = (int **)malloc(N * sizeof(int*));
    for(int i = 0; i < N; i++) B[i] = (int *)malloc(N * sizeof(int));
    C = (int **)malloc(N * sizeof(int*));
    for(int i = 0; i < N; i++) C[i] = (int *)malloc(N * sizeof(int));

    h_A = (int *)malloc(N*N*sizeof(int));
    h_B = (int *)malloc(N*N*sizeof(int));
    h_C = (int *)malloc(N*N*sizeof(int));

    error = hipMalloc(&d_A,N*N*sizeof(int));
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_A (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMalloc(&d_B,N*N*sizeof(int));
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_B (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMalloc(&d_C,N*N*sizeof(int));
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_C (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMalloc(&d_N,sizeof(int));
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_N (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMalloc(&d_H,sizeof(int));
     if (error != hipSuccess){
        fprintf(stderr, "Failed to allocate mem for d_H (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    /**************set initial values****************/

    // Read matrix A and B
    readMatrix(fileA, A, N);
    readMatrix(fileB, B, N);

    // write A and B on array
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            h_A[(i*N)+j]=A[i][j];
            h_B[(i*N)+j]=B[i][j];
        }
    }

    // Print matrix A and B
    printMatrix(A, N);
    printMatrix(B, N);
  
    //Memcpy: Host to device

    error = hipMemcpy(d_A, h_A, N*N*sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device(error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_B, h_B, N*N*sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device(error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_C, h_C, N*N*sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device(error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_H, &H, sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device(error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy on device(error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    //Blocks and threads definition
    int blocks = atoi(argv[5]);

    //Launch Kernel

    multiplyMat<<<blocks,H>>>(d_A,d_B, d_C, d_H, d_N);

    error = hipGetLastError();
    if (error != hipSuccess){
        fprintf(stderr, "Failed to launch multiplyMatrix (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }


    // Memcpy : Device to Host


    error = hipMemcpy(h_C, d_C, N*N*sizeof(int), hipMemcpyDeviceToHost);
     if (error != hipSuccess){
        fprintf(stderr, "Failed to  to copy from device (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            C[i][j]=h_C[(i*N)+j];
        }
    }

    //print results
    printMatrix(C, N);

    // Write the matrix
    writeMatrix(argv[6], C, N);
    
    // free memory
    
    hipFree(d_N);
    hipFree(d_H);
    hipFree(d_C);
    hipFree(d_A);
    hipFree(d_B);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}